#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include "utils.h"

extern __global__ void MatrixMultiply(float * mat_a, float * mat_b, float * mat_c, int m, int n, int k);
extern __global__ void MatrixMultiplySmem(float * mat_a, float * mat_b, float * mat_c, int m, int n, int k);
extern void MatrixMultiply_CPU_Native(float * mat_a, float * mat_b, float * mat_c, int m, int n, int k);
extern void MatrixMultiply_CPU_OPT1(float * mat_a, float * mat_b, float * mat_c, int m, int n, int k);
extern void MatrixMultiply_CPU_OPT2(float * mat_a, float * mat_b, float * mat_c, int m, int n, int k, int b);

__global__ void Print(void)
{
    printf("gridDim.x=%d, gridDim.y=%d\n", gridDim.x, gridDim.y);
    printf("blockDim.x=%d, blockDim.y=%d\n", blockDim.x, blockDim.y);
    int idx = threadIdx.x + blockDim.x*blockIdx.x;
    int idy = threadIdx.y + blockDim.y*blockIdx.y;
    //printf("%d, %d, %d, %d, %d, %d, %d, %d\n", idx, idy, threadIdx.x, threadIdx.y, blockDim.x, blockDim.y, blockIdx.x, blockIdx.y);
}

int main(int argc, char**argv)
{
    if (argc < 6) {
        printf("Usage: \n");
        printf("    matrix_multiply  m  n  k  b_h b_w\n");
    }
    int m = atoi(argv[1]);
    int n = atoi(argv[2]);
    int k = atoi(argv[3]);
    int b_h = atoi(argv[4]);
    int b_w = atoi(argv[5]); 
    printf("matrix A is [%d %d]\n", m, k);
    printf("matrix B is [%d %d]\n", k, n);
    printf("matrix C is [%d %d]\n", m, n);
    printf("block is [%d %d]\n", b_h, b_w);
    // initialize gpu device
    initDevice(0);

    // display gpu info
    gpu_info_display();
    
    
    int bytes_a = m*k*sizeof(float);
    int bytes_b = k*n*sizeof(float);
    int bytes_c = m*n*sizeof(float);

    //Malloc
    printf("Malloc memory on Host\n");
    float * a_host = (float*)malloc(bytes_a);
    if (a_host == NULL) {
        printf("matrix a_host malloc fail\n");
        return 0;
    }
    memset((void*)a_host, 0, bytes_a);
    float * b_host = (float*)malloc(bytes_b);
    if (b_host == NULL) {
        printf("matrix b_host malloc fail\n");
        free(a_host);
        return 0;
    }
    memset((void*)b_host, 0, bytes_b);
    float * c_host = (float*)malloc(bytes_c*5);
    if (c_host == NULL) {
        printf("matrix c_host malloc fail\n");
        free(b_host);
        free(a_host);
        return 0;
    }
    memset((void*)c_host, 0, bytes_c*5);
    float * c_host_opt1 = (float*)((char*)c_host + bytes_c);
    float * c_host_opt2 = (float*)((char*)c_host + bytes_c*2);
    float * c_from_dev  = (float*)((char*)c_host + bytes_c*3); 
    float * c_from_dev2 = (float*)((char*)c_host + bytes_c*4);

    // Initialize Data
    /*
    initialDataOne(a_host, bytes_a/sizeof(float));
    initialDataOne(b_host, bytes_b/sizeof(float));
    */
    initialDataRand(a_host, bytes_a/sizeof(float));
    initialDataRand(b_host, bytes_b/sizeof(float));
    
    printf("Malloc memory on Device\n");
    // Cuda Malloc
    float *a_dev=NULL;
    float *b_dev=NULL;
    float *c_dev=NULL;
    float *c_dev2=NULL;
    CHECK(hipMalloc((void**)&a_dev, bytes_a));
    CHECK(hipMalloc((void**)&b_dev, bytes_b));
    CHECK(hipMalloc((void**)&c_dev, bytes_c));
    CHECK(hipMalloc((void**)&c_dev2, bytes_c));


    CHECK(hipMemcpy(a_dev, a_host, bytes_a, hipMemcpyHostToDevice));
    CHECK(hipMemcpy(b_dev, b_host, bytes_b, hipMemcpyHostToDevice));  
    CHECK(hipMemset(c_dev, 0, bytes_c));
    CHECK(hipMemset(c_dev2, 0, bytes_c));

    // record time
    long long s_t;
    long long e_t;

    printf("CPU Native Execution...\n");
    // CPU matrix multiply native
    s_t = cpu_msec();
    MatrixMultiply_CPU_Native(a_host, b_host, c_host, m, n, k);
    e_t = cpu_msec();
    printf("CPU Native Execution Time elapsed %llu msec\n", e_t - s_t);
    // matrix_print(c_host, m, n);

    printf("CPU OPT1 Execution...\n");
    // CPU matrix multiply op1
    s_t = cpu_msec();
    MatrixMultiply_CPU_OPT1(a_host, b_host, c_host_opt1, m, n, k);
    e_t = cpu_msec();
    if (is_matrix_equal(c_host, c_host_opt1, m, n)) {
        printf("CPU OPT1 Execution Time elapsed %llu msec\n", e_t - s_t);
    }
    
    printf("CPU OPT2 Execution...\n");
    // CPU matrix multiply op2
    s_t = cpu_msec();
    MatrixMultiply_CPU_OPT2(a_host, b_host, c_host_opt2, m, n, k, 32);
    e_t = cpu_msec();
    if (is_matrix_equal(c_host, c_host_opt2, m, n)) {
        printf("CPU OPT2 Execution Time elapsed %llu msec\n", e_t - s_t);
    }

    // CUDA matrix_multiply
    dim3 block(b_h, b_w);
    dim3 grid((m-1)/b_h+1, (n-1)/b_w+1);

    printf("GPU Execution...\n");
    printf("Kernel configuration<<<(%d,%d), (%d,%d)>>>\n", grid.x, grid.y, block.x, block.y);
    s_t = cpu_msec();
    MatrixMultiply<<<grid, block>>>(a_dev, b_dev, c_dev, m, n, k);
    //Print<<<grid, block>>>();
    //hipError_t cudaStatus = hipGetLastError();
    //printf("CUDA error code=%d, reason=%s", cudaStatus, hipGetErrorString(cudaStatus));
    CHECK(hipDeviceSynchronize());
    e_t = cpu_msec();
    
    CHECK(hipMemcpy(c_from_dev, c_dev, bytes_c, hipMemcpyDeviceToHost));
    if (is_matrix_equal2(c_host, c_from_dev, m, n)) {
        printf("GPU Execution Time elapsed %llu msec\n", e_t - s_t);
    }

    printf("GPU Smem Execution...\n");
    printf("Kernel configuration<<<(%d,%d), (%d,%d)>>>\n", grid.x, grid.y, block.x, block.y);
    s_t = cpu_msec();
    MatrixMultiplySmem<<<grid, block>>>(a_dev, b_dev, c_dev2, m, n, k);
    //Print<<<grid, block>>>();
    //hipError_t cudaStatus = hipGetLastError();
    //printf("CUDA error code=%d, reason=%s", cudaStatus, hipGetErrorString(cudaStatus));
    CHECK(hipDeviceSynchronize());
    e_t = cpu_msec();

    CHECK(hipMemcpy(c_from_dev2, c_dev2, bytes_c, hipMemcpyDeviceToHost));
    if (is_matrix_equal2(c_from_dev, c_from_dev2, m, n)) {
        printf("GPU Smem Execution Time elapsed %llu msec\n", e_t - s_t);
    }


    hipFree(a_dev);
    hipFree(b_dev);
    hipFree(c_dev);
    free(a_host);
    free(b_host);
    free(c_host);

    hipDeviceReset();
    return 0;
}
